#include "hip/hip_runtime.h"
#include "device.cuh"
#include <memory>
int main()
{
    DataSet<float> dt("/home/u20/dataset/paper/f3_long_office");
    ck(hipGetLastError());

    cv::viz::Viz3d viz1("viz1"), viz2("viz2");
    viz1.showWidget("Coordinate", cv::viz::WCoordinateSystem());

    TSDF tsdf(make_float3(512, 512, 512), make_int2(640, 480));
    tsdf.pintr = new Intr(make_float4(550, 550, 320, 240));
    for (int i = 0; i < dt.pose.frames; i++)
    {
        cv::Mat rgb = cv::imread(dt.color_path[i]);
        cv::Mat depth = cv::imread(dt.depth_path[i], cv::IMREAD_ANYDEPTH);

        tsdf.addScan(depth, rgb);
        std::shared_ptr<Mat> cpu_cloud2;
        Mat cpu_color;
        tsdf.exportCloud(cpu_cloud2, cpu_color);
        viz1.showWidget("depthmode", cv::viz::WCloud(*cpu_cloud2, cpu_color));

        Mat depth_color, cpu_cloud;
        tsdf.depth2cam(depth, rgb, depth_color, cpu_color, cv::Affine3f::Identity());
        cv::Affine3f viewpose = cv::Affine3f::Identity();
        viz1.showWidget("depth", cv::viz::WCloud(depth_color, cpu_color), viewpose.translate(cv::Vec3f(4, 0, 0)));

        // cv::imshow("rgb", rgb);
        // cv::imshow("depth", depth);
        // cv::waitKey(100);
        viz1.spin();
    }
}