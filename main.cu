#include "hip/hip_runtime.h"
#include "device.cuh"
// #include <memory>

int main()
{
    loguru::g_stderr_verbosity = 9; // print everything

    DataSet<float> dt("/home/u20/dataset/paper/f3_long_office");
    ck(hipGetLastError());

    cv::viz::Viz3d viz1("viz1"), viz2("viz2");
    viz1.showWidget("Coordinate", cv::viz::WCoordinateSystem());

    TSDF tsdf(make_uint3(512, 512, 512), make_int2(640, 480));
    tsdf.pintr = new Intr(make_float4(550, 550, 320, 240));
    for (int i = 1; i < dt.pose.frames; i++)
    {
        // i = 1;
        cv::Mat rgb = cv::imread(dt.color_path[i]);
        cv::Mat depth = cv::imread(dt.depth_path[i], cv::IMREAD_ANYDEPTH);
        cv::Affine3f pose = dt.pose.getvectorPose(i);
        tsdf.addScan(depth, rgb,pose);
        // if (i <100)
        //     continue;
        Mat cpu_cloud2;
        Mat cpu_color;
        tsdf.exportCloud(cpu_cloud2, cpu_color);
        // viz1.showWidget("depthmode", cv::viz::WCloud(cpu_cloud2, cpu_color));
        viz1.showWidget("depthmode222", cv::viz::WCloud(cpu_cloud2, cpu_color));
        // Mat depth_color, cpu_cloud,cpu_color;
        // tsdf.depth2cam(depth, rgb, depth_color, cpu_color, cv::Affine3f::Identity());
        // cv::Affine3f viewpose = cv::Affine3f::Identity();
        // viz1.showWidget("depth", cv::viz::WCloud(depth_color, cpu_color), viewpose.translate(cv::Vec3f(4, 0, 0)));

        cv::imshow("rgb", rgb);
        cv::imshow("depth", depth);
        cv::waitKey(10);
        // viz1.spin();
        viz1.spinOnce(true);
    }
}